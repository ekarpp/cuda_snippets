#include "radix_sort.cu"

#include <vector>
#include <iostream>

std::vector<u64> random_u64(uint len)
{
    std::vector<u64> data(len);

    for (uint i = 0; i < len; i++)
        data[i] = ((u64) rand() << 32) | rand();

    return data;
}

std::vector<u32> random_u32(uint len)
{
    std::vector<u32> data(len);

    for (uint i = 0; i < len; i++)
        data[i] = rand() % 0xFF;

    return data;
}


bool is_sorted(u64 *vec, uint len)
{
    u64 prev = vec[0];

    for (uint i = 1; i < len; i++)
    {
        if (vec[i] > prev)
        {
            return false;
        }
        else
        {
            prev = vec[i];
        }
    }
    return true;
}

static void test_sort(uint len)
{
    std::cout << "Testing sort for " << len << " elements... ";

    std::vector<u64> input = random_u64(len);

    radix_sort(len, input.data());

    if (!is_sorted(input.data(), len))
    {
        std::cout << "FAIL";
    }
    else
    {
        std::cout << "OK";
    }

    std::cout << std::endl;
}

static void test_local_scan()
{
    std::cout << "Testing local scan... ";
    std::vector<u32> data = random_u32(ELEM_PER_BLOCK);
    u32* gpu = NULL;
    hipMalloc((void **) &gpu, data.size() * sizeof(u32));
    hipMemcpy(gpu, data.data(), data.size() * sizeof(u32), hipMemcpyHostToDevice);

    scan_histograms<false, false>
        <<<1, THREADS>>>
        (gpu, NULL);
    check_gpu_error("scan_histograms");

    std::vector<u32> out(data.size());
    hipMemcpy(out.data(), gpu, data.size() * sizeof(u32), hipMemcpyDeviceToHost);

    u32 sum = 0;

    for (uint i = 0; i < data.size(); i++)
    {
        if (sum != out[i])
        {
            std::cout << "FAIL at " << i << "/" << data.size() << std::endl;
            return;
        }
        sum += data[i];
    }

    std::cout << "OK" << std::endl;
}

static void test_global_scan()
{
    std::cout << "Testing global scan... ";
    const int scan_depth = 1;
    const int blocks = ELEM_PER_BLOCK;
    std::vector<u32> data = random_u32(blocks * blocks);
    int scan_sizes[1];
    scan_sizes[0] = blocks;

    u32 *gpu = NULL;
    hipMalloc((void **) &gpu, blocks * blocks * sizeof(u32));
    hipMemcpy(gpu, data.data(), blocks * blocks * sizeof(u32), hipMemcpyHostToDevice);

    u32 *scan_sums[1];
    hipMalloc((void **) &scan_sums[0], blocks * sizeof(u32));

    global_scan(gpu, scan_sums, scan_sizes, scan_depth, blocks);

    std::vector<u32> out(blocks * blocks);
    hipMemcpy(out.data(), gpu, blocks * blocks * sizeof(u32), hipMemcpyDeviceToHost);

    u32 sum = 0;
    for (uint i = 0; i < data.size(); i++)
    {
        sum += data[i];
        if (sum != out[i])
        {
            std::cout << "FAIL at " << i << "/" << data.size() << std::endl;
            return;
        }

    }

    std::cout << "OK" << std::endl;
}

static void test_sort_block()
{
    std::cout << "Testing sort block...";
    const int blocks = ELEM_PER_BLOCK;
    std::vector<u64> data = random_u64(blocks * blocks);
    for (int i = 0; i < data.size(); i++)
        data[i] &= 0xF;

    u64 *gpu = NULL;
    hipMalloc((void **) &gpu, blocks * blocks * sizeof(u64));
    hipMemcpy(gpu, data.data(), blocks * blocks * sizeof(u64), hipMemcpyHostToDevice);

    u64 *out = NULL;
    hipMalloc((void **) &out, blocks * blocks * sizeof(u64));

    sort_block
        <<<blocks, THREADS>>>
        ((u64_vec *) gpu, (u64_vec *) out, 0);

    u64 *sorted = (u64 *) std::malloc(blocks * blocks * sizeof(u64));
    hipMemcpy(sorted, out, blocks * blocks * sizeof(u64), hipMemcpyDeviceToHost);

    u64 offset = 0;
    while (offset < blocks)
    {
        if (!is_sorted(sorted + offset * blocks, blocks))
        {
            std::cout << "FAIL" << std::endl;
            return;
        }
        offset += blocks;
    }

    std::cout << "OK" << std::endl;
}

static void test_create_histogram()
{
    std::cout << "Testing create histogram...";
    const int blocks = ELEM_PER_BLOCK;

    std::vector<u64> data = random_u64(blocks * blocks);
    for (int i = 0; i < data.size(); i++)
        data[i] &= 0xF;

    u64 *gpu = NULL;
    hipMalloc((void **) &gpu, blocks * blocks * sizeof(u64));
    hipMemcpy(gpu, data.data(), blocks * blocks * sizeof(u64), hipMemcpyHostToDevice);

    u32 *grams = NULL;
    hipMalloc((void **) &grams, blocks * RADIX_SIZE * sizeof(u32));

    u32 *start_ptrs = NULL;
    hipMalloc((void **) &start_ptrs, blocks * RADIX_SIZE * sizeof(u32));

    compute_histograms
        <<<blocks, THREADS>>>
        ((u64_vec *) gpu, grams, start_ptrs, blocks, 0);

    std::vector<u32> out(blocks * RADIX_SIZE);
    hipMemcpy(out.data(), grams, blocks * RADIX_SIZE * sizeof(u32), hipMemcpyDeviceToHost);

    for (int i = 0; i < blocks; i++)
    {
        std::vector<int> local(RADIX_SIZE, 0);

        for (int j = 0; j < ELEM_PER_BLOCK; j++)
            local[data[i * blocks + j]]++;

        for (int j = 0; j < RADIX_SIZE; j++)
        {
            if (local[j] != out[j * blocks + i])
            {
                std::cout << "FAIL" << std::endl;
                return;
            }
        }
    }

    std::cout << "OK" << std::endl;
}

int main()
{
    test_sort_block();
    test_local_scan();
    test_global_scan();
    test_create_histogram();
    test_sort(1024);
    test_sort(1024 * 1024);
    return 0;
}
